#include "sketch/hllgpu.h"
#include <cstdio>
#include "omp.h"

using namespace sketch;
int main(int argc, char *argv[]) {
    size_t n = argc == 1 ? 60: std::atoi(argv[1]);
    int p = argc <= 2 ? 14: std::atoi(argv[2]);
    omp_set_num_threads(24);
//__host__ std::vector<float> all_pairs(const uint8_t *p, unsigned l2, size_t nhlls) {
    std::vector<hll::hll_t> hlls;
    hlls.reserve(n);
    for(int i = 0; i < n; ++i) hlls.emplace_back(p);
    OMP_PRAGMA("omp parallel for")
    for(size_t i = 0; i < 10000000;  ++i) {
        if(i % 100000 == 0) std::fprintf(stderr, "tid %d with %zu\n", omp_get_thread_num(), i);
        if(i & 1) {
            for(auto &h: hlls) h.addh(i);
        } else if(i & 2) {
            for(int j = 0; j < n; j += 2)
                hlls[j].addh(i);
        } else {
            for(int j = i % 7; j < n; j = j + 3)
                hlls[j].addh(i);
        }
    }
    std::fprintf(stderr, "Finished making now copy\n");
    std::vector<float> cards;
    cards.reserve(hlls.size());
    int mgs, bs, gs;
    hipError_t ce;
    if((ce = hipOccupancyMaxPotentialBlockSize(&mgs, &bs, calc_sizes_large, 0, 0)))
        throw CudaError(ce, "Failed to infer best block size and so on.");
    gs = (hlls.size() * hlls.size() / 2 + (bs - 1)) / bs;
    std::fprintf(stderr, "mgs: %d. bs: %d. gs: %d\n", mgs, bs, gs);
    for(auto &h: hlls) cards.push_back(h.report());
    for(size_t i = 0; i < hlls.size(); std::fprintf(stderr, "size: %lf\n", hlls[i++].report()));
    std::vector<uint8_t> cd(n << p);
    for(size_t i = 0; i < n; ++i) {
        std::memcpy(cd.data() + (i << p), hlls[i].data(), size_t(1) << p);
    }
    uint8_t *ddata;
    if(hipMalloc((void **)&ddata, (n << p))) throw std::runtime_error("Failed to allocate on device");
    if(hipMemcpy(ddata, cd.data(), n << p, hipMemcpyHostToDevice)) throw std::runtime_error("Failed to copy to device");
    std::fprintf(stderr, "Finish copy\n");
    size_t time;
    auto sizes = all_pairsu(ddata, p, n, time);
    auto s2 = std::vector<uint32_t>(sizes.size());
    auto t = hrc::now();
    for(auto i = 0u; i < hlls.size(); ++i) {
        OMP_PRAGMA("omp parallel for")
        for(auto j = i + 1; j < hlls.size(); ++j) {
            s2[ij2ind(i, j, hlls.size())] = jaccard_index(hlls[i], hlls[j]);
        }
    }
    auto t2 = hrc::now();
    size_t time2 = (t2 - t).count();
    std::fprintf(stderr, "time diff: %zu\n", time2);
    std::fprintf(stderr, "time ratio: %lf\n", double(time2) / time);
    hipFree(ddata);
}
